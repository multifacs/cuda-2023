
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <omp.h>
#include <stdio.h>

#include <iostream>

__global__ void saxpy_kernel(const int n, const float a, float *x,
                             const int incx, float *y, const int incy) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    y[i * incy] += a * x[i * incx];
  }
}

__global__ void daxpy_kernel(const int n, const double a, double *x,
                             const int incx, double *y, const int incy) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    y[i * incy] += a * x[i * incx];
  }
}

void saxpy_gpu(const int n, const float a, float *x, const int incx, float *y,
               const int incy, const int numBlocks, const int blocksSize) {
  hipError_t cudaStatus;
  int sizeX = 1 + (n - 1) * abs(incx);
  int sizeY = 1 + (n - 1) * abs(incy);

  float *gpuX;
  cudaStatus = hipMalloc((void **)&gpuX, sizeX * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "cudaMalloc(gpuX) faild f1\n");
    return;
  }

  float *gpuY;
  cudaStatus = hipMalloc((void **)&gpuY, sizeY * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "cudaMalloc(gpuY) faild f2\n");
    return;
  }

  cudaStatus =
      hipMemcpy(gpuX, x, sizeX * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "cudaMemcpy(gpuX) faild\n");
    return;
  }

  cudaStatus =
      hipMemcpy(gpuY, y, sizeY * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "cudaMemcpy(gpuY) faild\n");
    return;
  }

  hipEvent_t startTime, stopF;
  float gpuTimeF = 0.0f;

  hipEventCreate(&startTime);
  hipEventCreate(&stopF);
  hipEventRecord(startTime, 0);

  saxpy_kernel<<<numBlocks, blocksSize>>>(n, a, gpuX, incx, gpuY, incy);

  hipEventRecord(stopF, 0);
  hipEventSynchronize(stopF);
  hipEventElapsedTime(&gpuTimeF, startTime, stopF);

  printf("OpenGL: %dms  size: %d", int(gpuTimeF), blocksSize);

  cudaStatus =
      hipMemcpy(y, gpuY, sizeY * sizeof(float), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "cudaMemcpy(gpuY) faild\n");
    return;
  }

  hipFree(gpuX);
  hipFree(gpuY);
  hipEventDestroy(startTime);
  hipEventDestroy(stopF);
  return;
}

void daxpy_gpu(const int n, const double a, double *x, const int incx,
               double *y, const int incy, const int numBlocks,
               const int blocksSize) {
  hipError_t cudaStatus;
  int sizeX = 1 + (n - 1) * abs(incx);
  int sizeY = 1 + (n - 1) * abs(incy);

  double *gpuX;
  cudaStatus = hipMalloc((void **)&gpuX, sizeX * sizeof(double));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "cudaMalloc(gpuX) faild d1\n");
    return;
  }

  double *gpuY;
  cudaStatus = hipMalloc((void **)&gpuY, sizeY * sizeof(double));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "cudaMalloc(gpuY) faild d2\n");
    return;
  }

  cudaStatus =
      hipMemcpy(gpuX, x, sizeX * sizeof(double), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "cudaMemcpy(gpuX) faild\n");
    return;
  }

  cudaStatus =
      hipMemcpy(gpuY, y, sizeY * sizeof(double), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "cudaMemcpy(gpuY) faild\n");
    return;
  }

  hipEvent_t startTime, stopD;
  float gpuTimeD = 0.0f;

  hipEventCreate(&startTime);
  hipEventCreate(&stopD);
  hipEventRecord(startTime, 0);

  daxpy_kernel<<<numBlocks, blocksSize>>>(n, a, gpuX, incx, gpuY, incy);

  hipEventRecord(stopD, 0);
  hipEventSynchronize(stopD);
  hipEventElapsedTime(&gpuTimeD, startTime, stopD);

  printf("OpenGL: %dms  size: %d", int(gpuTimeD), blocksSize);

  cudaStatus =
      hipMemcpy(y, gpuY, sizeY * sizeof(double), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "cudaMemcpy(gpuY) faild\n");
    return;
  }

  hipFree(gpuX);
  hipFree(gpuY);
  hipEventDestroy(startTime);
  hipEventDestroy(stopD);
  return;
}

template <typename t>
bool comp(t *a1, t *a2, size_t size) {
  for (size_t i = 0; i < size; i++) {
    if (a1[i] != a2[i]) return false;
  }
  return true;
}

void saxpy(const int n, const float a, float *x, const int incx, float *y,
           const int incy) {
  const int biasx = incx < 0 ? (n - 1) * abs(incx) : 0;
  const int biasy = incy < 0 ? (n - 1) * abs(incy) : 0;

  for (size_t i = 0; i < n; i++) {
    y[biasy + i * incy] += a * x[biasx + i * incx];
  }
}

void daxpy(const int n, const double a, double *x, const int incx, double *y,
           const int incy) {
  const int biasx = incx < 0 ? (n - 1) * abs(incx) : 0;
  const int biasy = incy < 0 ? (n - 1) * abs(incy) : 0;

  for (size_t i = 0; i < n; i++) {
    y[biasy + i * incy] += a * x[biasx + i * incx];
  }
}

void saxpy_omp(const int n, const float a, float *x, const int incx, float *y,
               const int incy) {
  const int biasx = incx < 0 ? (n - 1) * abs(incx) : 0;
  const int biasy = incy < 0 ? (n - 1) * abs(incy) : 0;

#pragma omp parallel for num_threads(4)
  for (int i = 0; i < n; i++) {
    y[biasy + i * incy] += a * x[biasx + i * incx];
  }
}

void daxpy_omp(const int n, const double a, double *x, const int incx,
               double *y, const int incy) {
  const int biasx = incx < 0 ? (n - 1) * abs(incx) : 0;
  const int biasy = incy < 0 ? (n - 1) * abs(incy) : 0;

#pragma omp parallel for num_threads(4)
  for (int i = 0; i < n; i++) {
    y[biasy + i * incy] += a * x[biasx + i * incx];
  }
}

int main() {
  const int n = 50000000;  // 1e7;
  const int incx = 10;
  const int incy = 10;
  const int sizeX = 1 + (n - 1) * abs(incx);
  const int sizeY = 1 + (n - 1) * abs(incy);
  int block_size;
  int num_blocks;

  const float aFloat = 10.0f;
  float *xFloat = new float[sizeX];
  float *yFloat = new float[sizeY];
  for (int i = 0; i < n; ++i) {
    xFloat[i] = 5.0f;
    yFloat[i] = 1.0f;
  }

  double startTime = omp_get_wtime();
  saxpy(n, aFloat, xFloat, incx, yFloat, incy);
  double endTime = omp_get_wtime();

  printf("Saxpy Type Float\n");
  printf("Sequential: %dms", int((endTime - startTime) * 1000.0));
  printf("\n");

  delete[] xFloat;
  delete[] yFloat;

  xFloat = new float[sizeX];
  yFloat = new float[sizeY];
  for (int i = 0; i < n; ++i) {
    xFloat[i] = 5.0f;
    yFloat[i] = 1.0f;
  }

  startTime = omp_get_wtime();
  saxpy_omp(n, aFloat, xFloat, incx, yFloat, incy);
  endTime = omp_get_wtime();

  printf("OpenMP: %dms", int((endTime - startTime) * 1000.0));
  printf("\n");

  delete[] xFloat;
  delete[] yFloat;

  for (int i = 8; i <= 128; i *= 2) {
    block_size = i;
    num_blocks = (n + block_size - 1) / block_size;

    xFloat = new float[sizeX];
    yFloat = new float[sizeY];

    for (int i = 0; i < n; ++i) {
      xFloat[i] = 5.0;
      yFloat[i] = 1.0;
    }
    saxpy_gpu(n, aFloat, xFloat, incx, yFloat, incy, num_blocks, block_size);
    printf("\n");

    delete[] xFloat;
    delete[] yFloat;
  }

  const double aDouble = 10.0;
  double *xDouble = new double[sizeX];
  double *yDouble = new double[sizeY];
  for (int i = 0; i < n; ++i) {
    xDouble[i] = 5.0;
    yDouble[i] = 1.0;
  }

  startTime = omp_get_wtime();
  daxpy(n, aDouble, xDouble, incx, yDouble, incy);
  endTime = omp_get_wtime();

  printf("\n");
  printf("Daxpy Type Double\n");
  printf("Sequential: %dms", int((endTime - startTime) * 1000.0));
  printf("\n");

  delete[] xDouble;
  delete[] yDouble;

  xDouble = new double[sizeX];
  yDouble = new double[sizeY];
  for (int i = 0; i < n; ++i) {
    xDouble[i] = 5.0;
    yDouble[i] = 1.0;
  }

  startTime = omp_get_wtime();
  daxpy_omp(n, aDouble, xDouble, incx, yDouble, incy);
  endTime = omp_get_wtime();

  printf("OpenMP: %dms", int((endTime - startTime) * 1000));
  printf("\n");
  delete[] xDouble;
  delete[] yDouble;

  for (int i = 8; i <= 128; i *= 2) {
    block_size = i;
    num_blocks = (n + block_size - 1) / block_size;

    xDouble = new double[sizeX];
    yDouble = new double[sizeY];
    for (int i = 0; i < n; ++i) {
      xDouble[i] = 5.0;
      yDouble[i] = 1.0;
    }

    daxpy_gpu(n, aDouble, xDouble, incx, yDouble, incy, num_blocks, block_size);
    printf("\n");

    delete[] xDouble;
    delete[] yDouble;
  }

  return 0;
}