#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>

#include <iostream>

__global__ void float_matrix_multiplication_kernel(const int m, const int n,
                                                   const int k, float* x,
                                                   float* y, float* z) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < k && row < m) {
    for (int i = 0; i < n; i++) {
      z[row * k + col] += x[row * n + i] * y[i * k + col];
    }
  }
}

__global__ void block_float_matrix_multiplication_kernel(const int m,
                                                         const int n,
                                                         const int k, float* x,
                                                         float* y, float* z) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  float res = 0;
  __shared__ float a_block[16 * 16];
  __shared__ float b_block[16 * 16];
  if (col < k && row < m) {
    for (int i = 0; i < n; i += blockDim.y) {
      a_block[threadIdx.y * blockDim.x + threadIdx.x] =
          x[(blockIdx.y * blockDim.y + threadIdx.y) * n + (i + threadIdx.x)];
      b_block[threadIdx.y * blockDim.y + threadIdx.x] =
          y[(i + threadIdx.y) * k + (blockIdx.x * blockDim.x + threadIdx.x)];
      __syncthreads();
      for (int j = 0; j < blockDim.x; j++) {
        res += a_block[threadIdx.y * blockDim.y + j] *
               b_block[j * blockDim.y + threadIdx.x];
      }
      __syncthreads();
    }
    z[row * k + col] += res;
  }
}

void float_matrix_multiplication_cuda(const int m, const int n, const int k,
                                      const float* x, const float* y, float* z,
                                      const dim3 dimGrid, const dim3 dimBlock) {
  hipError_t cudaStatus;

  float *gpuX, *gpuY, *gpuZ;
  cudaStatus = hipMalloc((void**)&gpuX, n * m * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc(gpuX) faild\n");
    return;
  }
  cudaStatus = hipMalloc((void**)&gpuY, n * k * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc(gpuY) faild\n");
    return;
  }
  cudaStatus = hipMalloc((void**)&gpuZ, m * k * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc(gpuZ) faild\n");
    return;
  }

  cudaStatus =
      hipMemcpy(gpuX, x, n * m * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy(gpuX) faild\n");
    return;
  }

  cudaStatus =
      hipMemcpy(gpuY, y, n * k * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy(gpuY) faild\n");
    return;
  }

  cudaStatus =
      hipMemcpy(gpuZ, z, m * k * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy(gpuZ) faild\n");
    return;
  }

  hipEvent_t start, stop;
  float gpuTime = 0.0f;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  float_matrix_multiplication_kernel<<<dimGrid, dimBlock>>>(m, n, k, gpuX, gpuY,
                                                            gpuZ);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&gpuTime, start, stop);

  printf("CUDA: %dms", int(gpuTime));
  printf("\n");

  cudaStatus =
      hipMemcpy(z, gpuZ, m * k * sizeof(float), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy(gpuZ) faild\n");
    return;
  }

  hipFree(gpuX);
  hipFree(gpuY);
  hipFree(gpuZ);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return;
}

void block_float_matrix_multiplication_cuda(const int m, const int n,
                                            const int k, const float* x,
                                            const float* y, float* z,
                                            const dim3 dimGrid,
                                            const dim3 dimBlock) {
  hipError_t cudaStatus;

  float *gpuX, *gpuY, *gpuZ;
  cudaStatus = hipMalloc((void**)&gpuX, n * m * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc(gpuX) faild\n");
    return;
  }
  cudaStatus = hipMalloc((void**)&gpuY, n * k * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc(gpuY) faild\n");
    return;
  }
  cudaStatus = hipMalloc((void**)&gpuZ, m * k * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc(gpuZ) faild\n");
    return;
  }

  cudaStatus =
      hipMemcpy(gpuX, x, n * m * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy(gpuX) faild\n");
    return;
  }

  cudaStatus =
      hipMemcpy(gpuY, y, n * k * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy(gpuY) faild\n");
    return;
  }

  cudaStatus =
      hipMemcpy(gpuZ, z, m * k * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy(gpuZ) faild\n");
    return;
  }

  hipEvent_t start, stop;
  float gpuTime = 0.0f;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  block_float_matrix_multiplication_kernel<<<dimGrid, dimBlock>>>(m, n, k, gpuX,
                                                                  gpuY, gpuZ);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&gpuTime, start, stop);

  printf("CUDA GEMM: %dms", int(gpuTime));
  printf("\n");

  cudaStatus =
      hipMemcpy(z, gpuZ, m * k * sizeof(float), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy(gpuZ) faild\n");
    return;
  }

  hipFree(gpuX);
  hipFree(gpuY);
  hipFree(gpuZ);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return;
}

void float_matrix_multiplication(const int m, const int n, const int k,
                                 float* x, float* y, float* z) {
  for (int i = 0; i < m; ++i)
    for (int p = 0; p < k; ++p)
      for (int j = 0; j < n; ++j) z[i * k + p] += x[i * n + j] * y[j * k + p];
}

void float_matrix_multiplication_omp(const int m, const int n, const int k,
                                     float* x, float* y, float* z) {
  int i, p, j;
  omp_set_num_threads(2);
#pragma omp parallel for private(i, p, j) shared(x, y, z)
  for (i = 0; i < m; ++i) {
    for (p = 0; p < n; ++j) {
      float sum = 0;
      for (j = 0; j < k; ++k) {
        sum += x[i * n + j] * y[j * k + p];
      }
      z[i * k + p] = sum;
    }
  }
}

int main() {
  const int N = 1024;

  float* x = new float[N * N];
  float* y = new float[N * N];
  float* z = new float[N * N];

  for (int i = 0; i < N * N; i++) {
    x[i] = 1.0;
  }
  for (int i = 0; i < N * N; i++) {
    y[i] = 1.0;
  }
  for (int i = 0; i < N * N; i++) {
    z[i] = 0.0;
  }

  float startTime = omp_get_wtime();
  float_matrix_multiplication(N, N, N, x, y, z);
  float endTime = omp_get_wtime();

  printf("Sequential: %dms", int((endTime - startTime) * 1000));
  printf("\n");

  startTime = omp_get_wtime();
  float_matrix_multiplication_omp(N, N, N, x, y, z);
  endTime = omp_get_wtime();

  printf("OpenMP: %dms\n\n", int((endTime - startTime) * 1000));

  dim3 dimBlockf(16, 16);
  dim3 dimGridf((N + dimBlockf.x - 1) / dimBlockf.x,
                (N + dimBlockf.y - 1) / dimBlockf.y);

  float_matrix_multiplication_cuda(N, N, N, x, y, z, dimGridf, dimBlockf);

  dim3 dimBlock_1f(16, 16);
  dim3 dimGrid_1f((N + dimBlock_1f.x - 1) / dimBlock_1f.x,
                  (N + dimBlock_1f.y - 1) / dimBlock_1f.y);

  block_float_matrix_multiplication_cuda(N, N, N, x, y, z, dimGrid_1f,
                                         dimBlock_1f);

  delete[] x, delete[] y, delete[] z;

  return 0;
}